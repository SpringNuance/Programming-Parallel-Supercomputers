#include <stdlib.h>
#include <stdio.h>
#include <algorithm>
#include <hip/hip_runtime_api.h>
#if DISTRIBUTED
    #include "quicksort_distributed.cuh"
    #include <mpi.h>
#else
    #include "quicksort.cuh"
#endif
bool is_sorted(float* data, int n){
    for(int i=1;i<n;i++){
        if(data[i]<data[i-1]){
            return false;
        }
    }
    return true;
}

int main(int argc, char* argv[]){
    int rank;
    #if DISTRIBUTED
        MPI_Init(&argc, &argv);
        MPI_Comm_rank(MPI_COMM_WORLD, &rank); 
    #else
        rank = 0;
    #endif
    constexpr int size = 20000;
    bool multi = false;
    float data[size];
    float* c_data = (float*)malloc(size*sizeof(float));
    float* g_data;
    float* result;
    float data_gpu[size];

    srand(12345678);
    for(int i=0;i<size;i++){
        data[i] = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
    }
    #if GPU
        hipMalloc((void**)&g_data,size*sizeof(float));
        hipDeviceSynchronize();
       
        hipMemcpy(g_data,data,size*sizeof(float),hipMemcpyHostToDevice);
        hipDeviceSynchronize();
        result = data_gpu;
        #if DISTRIBUTED 
            quicksort_distributed(data[0],0,size,g_data,MPI_COMM_WORLD);
            //quicksort(data[0],0,size,g_data);
        #else
            quicksort(data[0],0,size,g_data);
        #endif
        hipDeviceSynchronize();
        hipMemcpy(data_gpu,g_data,size*sizeof(float),hipMemcpyDeviceToHost);
        hipDeviceSynchronize();
        hipFree(g_data);
    #else
        for(int i=0;i<size;i++) c_data[i] = data[i];
        #if DISTRIBUTED
            quicksort_distributed(data[0],0,size,c_data,MPI_COMM_WORLD);
        #else
            quicksort(data[0],0,size,c_data);
        #endif
        result = c_data;
    #endif
    
    std::stable_sort(data,data+size);
    bool sorted = is_sorted(result,size);
    if(sorted){
        printf("Is sorted at rank %d\n", rank);
    }else{
        printf("Not sorted at rank %d!!\n", rank);
    }
    bool is_correct = true;
    for(int i=0;i<size;i++) is_correct &= (result[i] == data[i]);
    if(is_correct){
        printf("Correct at rank: %d\n",rank);
    }else{
        printf("Incorrect at rank %d!!\n",rank);
    }
    #if DISTRIBUTED
	MPI_Barrier(MPI_COMM_WORLD);
        MPI_Finalize();
    #endif
    return 0;

}

