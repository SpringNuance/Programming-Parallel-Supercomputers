// Overlapping data transfers and communications example.


#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ 
void compute_kernel(float *d_A, int offset)
{
  int i = offset + threadIdx.x + blockIdx.x*blockDim.x;
  float x = (float)i;
  float y = (float)gridDim.x;
  d_A[i]=1.-2.*(x/y)*(x/y) + (x/y)*(x/y)*(x/y); 
}

int main(int argc, char **argv)
{
  const int blockSize = 256, nStreams = 4;
  const int n = 4 * 1024 * blockSize * nStreams;
  const int streamSize = n / nStreams;
  const int streamBytes = streamSize * sizeof(float);
  const int bytes = n * sizeof(float);
   
  //Host memory; reserved in a "normal way".
  //If one does the mem alloc for host this way, the blocking scheme is faster
  //float *h_A = (float*)malloc(bytes);
  //Using pinned memory streams outperform in transfer speed; read more from
  //https://developer.nvidia.com/blog/how-optimize-data-transfers-cuda-cc/
  float *h_A;
  hipHostMalloc((void**)&h_A, bytes, hipHostMallocDefault) ;
  // End pinned
  float *d_A;
  hipMalloc((void**)&d_A, bytes) ; // device

  float timems; // elapsed time in milliseconds
  
  // create events and streams
  hipEvent_t startEvent, stopEvent, dummyEvent;
  hipStream_t stream[nStreams];
  hipEventCreate(&startEvent) ;
  hipEventCreate(&stopEvent) ;
  hipEventCreate(&dummyEvent) ;
  for (int i = 0; i < nStreams; ++i)
    hipStreamCreate(&stream[i]);
  
  // On one stream transfers should be blocking, and sequentialise the code.
  memset(h_A, 0, bytes);
  hipEventRecord(startEvent,0);
  hipMemcpy(d_A, h_A, bytes, hipMemcpyHostToDevice) ;
  compute_kernel<<<n/blockSize, blockSize>>>(d_A, 0);
  hipMemcpy(h_A, d_A, bytes, hipMemcpyDeviceToHost) ;
  hipEventRecord(stopEvent, 0) ;
  hipEventSynchronize(stopEvent) ;
  hipEventElapsedTime(&timems, startEvent, stopEvent) ;
  printf("Time for blocking transfers and computation (ms): %f\n", timems);

  // Let us try the same on streams, when transfers and computation should become concurrent and therefore more efficient
  memset(h_A, 0, bytes);
  hipEventRecord(startEvent,0);
  for (int i = 0; i < nStreams; ++i) {
    int offset = i * streamSize;
    hipMemcpyAsync(&d_A[offset], &h_A[offset], 
                               streamBytes, hipMemcpyHostToDevice, 
                               stream[i]);
    compute_kernel<<<streamSize/blockSize, blockSize, 0, stream[i]>>>(d_A, offset);
    hipMemcpyAsync(&h_A[offset], &d_A[offset], 
                               streamBytes, hipMemcpyDeviceToHost,
                               stream[i]);
  }
  hipEventRecord(stopEvent, 0);
  hipEventSynchronize(stopEvent);
  hipEventElapsedTime(&timems, startEvent, stopEvent);
  printf("Time for transfer and execution on %d streams (ms): %f\n", nStreams,timems);

  // Deallocate
  hipEventDestroy(startEvent) ;
  hipEventDestroy(stopEvent) ;
  hipEventDestroy(dummyEvent) ;
  for (int i = 0; i < nStreams; ++i)
    hipStreamDestroy(stream[i]) ;
  hipFree(d_A);
  hipHostFree(h_A);

  return 0;
}
