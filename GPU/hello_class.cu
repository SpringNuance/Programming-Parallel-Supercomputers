#include <mpi.h>
#include <stdio.h>
#include <stdlib.h>


int
main(void)
{
// Check how many MPIs we have
  MPI_Init(NULL, NULL);
  int nprocs, pid;
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
  printf("Num MPI processes: %d\n", nprocs);


// Check how many GPUs we have
  int num_devices;
  hipGetDeviceCount(&num_devices);
  printf("Num devices: %d\n", num_devices);

}

