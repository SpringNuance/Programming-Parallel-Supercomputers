#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#if USE_MPI
#include <mpi.h>
#endif

#include "errchk.h"
#include "reduce.cuh"

#define NUM_SAMPLES (512)
#define MAX_COUNT ((1 * 1024 * 1024) / sizeof(int))
#define ARRAY_SIZE(arr) (sizeof(arr) / sizeof(arr[0]))

static int
modelreduce(const int* arr, const size_t count)
{
  ERRCHK(count > 0);
  int max = arr[0];

  for (int i = 1; i < count; ++i)
    max = arr[i] > max ? arr[i] : max;

  return max;
}

static void
write_to_file(const char* path, const char* str)
{
  FILE* fp = fopen(path, "w");
  ERRCHK(fp);

  fprintf(fp, "%s\n", str);

  fclose(fp);
}

int
main(int argc, char* argv[])
{
  char* outfile = NULL;
  if (argc > 1)
    outfile = argv[1];

#if USE_MPI
  MPI_Init(NULL, NULL);
  int nprocs, pid;
  MPI_Comm_size(MPI_COMM_WORLD, &nprocs);
  MPI_Comm_rank(MPI_COMM_WORLD, &pid);
#else
  int num_devices;
  ERRCHK_CUDA(hipGetDeviceCount(&num_devices));
  const int nprocs = num_devices;
  const int pid    = 0;
#endif

  srand(982357885u);
  for (size_t i = 0; i < NUM_SAMPLES; ++i) {
    // Set count a multiple of nprocs/devices for simplicity
    const size_t count = nprocs * (1 + (rand() % (MAX_COUNT / nprocs)));
    int* arr           = (int*)malloc(count * sizeof(arr[0]));
    for (size_t i = 0; i < count; ++i)
      arr[i] = rand();

    const int model = modelreduce(arr, count);

#if USE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif
    const int candidate = reduce(arr, count);
#if USE_MPI
    MPI_Barrier(MPI_COMM_WORLD);
#endif

    int failed = 0;
    if (pid == 0) {
      printf("Model: %d, Candidate: %d, Correct? %s\n", model, candidate,
             model == candidate ? "Yes" : "No");
      fflush(stdout);

      if (model != candidate) {
        fprintf(stderr,
                "Failure:  Model: %d, Candidate: %d, "
                "Correct? %s\n",
                model, candidate, model == candidate ? "Yes" : "No");
        if (outfile)
          write_to_file(outfile, "FAILURE");

        failed = 1;
      }
    }
    free(arr);

#if USE_MPI
    MPI_Bcast(&failed, 1, MPI_INT, 0, MPI_COMM_WORLD);
#endif
    if (failed) {
      if (pid == 0 && outfile)
        write_to_file(outfile, "FAILURE");

#if USE_MPI
      MPI_Finalize();
#endif
      printf("pid %d exit failure\n", pid);
      return EXIT_FAILURE;
    }
  }

#if USE_MPI
  MPI_Finalize();
#endif

  if (outfile)
    write_to_file(outfile, "OK");

  printf("pid %d exit success\n", pid);
  return EXIT_SUCCESS;
}